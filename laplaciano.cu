
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include<stdio.h>
#include <algorithm>

#define BLOCK_SIZE 16
//int const Nx = 30, Nz = 20;

__global__
void laplacian_GPU (int ordem, int Nz, int Nx,int dz, int dx, float *P, float *Lapla)
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int colStride = blockDim.x * gridDim.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int rowStride = blockDim.y * gridDim.y;

    int in_n = ordem / 2;
    int lim_nx = Nx - ordem / 2;
    int lim_nz = Nz - ordem / 2;

    float Pxx,Pzz;

//    printf("in_n = %d",in_n);
    printf("lim_nx = %d",lim_nx);
    for (int i = col; i < lim_nz && i >= in_n; i += colStride){
        for (int j = row; j < lim_nx && j >= in_n; j += rowStride){
            Pzz = P[(i+1) * Nx + j] + P[(i-1) * Nx + j] - P[i * Nx + j] * 2.0;
            Pxx = P[i * Nx + j+1] + P[i * Nx + j-1] - P[i * Nx + j] * 2.0;
            Lapla[i * Nx + j] = Pxx/(dx*dx) + Pzz/(dz*dz);
            printf("Pxx %.3f, Pzz %.3f,i %d, j %d\n",Pxx,Pzz,i,j);
        }
    }
}

void laplacian(int Nx,int Nz,float dx,float dz)
{
    float **P = new float*[Nx];
    P[0] = new float[Nz * Nx];
    for (int i = 1; i < Nz; ++i)
        P[i] = P[i-1] + Nx;

    float **Lapla = new float*[Nx];
    Lapla[0] = new float[Nz * Nx];
    for (int i = 1; i < Nz; ++i)
        Lapla[i] = Lapla[i-1] + Nx;
    float *dP; hipMalloc((void **) &dP, sizeof(float) * Nz * Nx);
    float *dLapla; hipMalloc((void **) &dLapla, sizeof(float) * Nz * Nx);

    for (int i = 0; i < Nz; i++){
        for (int j = 0; j < Nx; j++){
            P[i][j] = i*j + i*i*i;
        };
    };

    hipMemcpy(dP, P[0],sizeof(float) * Nz * Nx, hipMemcpyHostToDevice);

    dim3 dimBlock(BLOCK_SIZE,BLOCK_SIZE);
    int Gridx = (Nx + dimBlock.x) / dimBlock.x;
    int Gridy = (Nz + dimBlock.y) / dimBlock.y;
    dim3 dimGrid(Gridx,Gridy);

    std::cout<<"dimBlock.x "<<dimBlock.x<<" dimBlock.y "<<dimBlock.y<<std::endl;
    std::cout<<"dimGrid.x "<<dimGrid.x<<" dimGrid.y "<<dimGrid.y<<std::endl;

    int ordem=2;

    laplacian_GPU<<<dimBlock,dimGrid>>>(ordem,Nz,Nx,dz,dx,dP,dLapla);

    hipError_t err = hipGetLastError();
    if(err != hipSuccess)
        std::cout<<"Error: "<<hipGetErrorString(err)<<std::endl;
    hipDeviceSynchronize();

    hipMemcpy(Lapla[0], dLapla,sizeof(float) * Nz * Nx, hipMemcpyDeviceToHost);

    for(int i=0; i<Nz; i++){
        for(int j=0; j<Nx; j++){
            std::cout << P[i][j] << "  ";
        }
        std::cout << std::endl;
    }

    for(int i=0; i<Nz; i++){
        for(int j=0; j<Nx; j++){
            std::cout << Lapla[i][j] << "  ";
        }
        std::cout << std::endl;
    }

    hipFree(P);
    hipFree(Lapla);
}

int main()
{
    int Nx = 10;
    int Nz = 10;
    float dz = 1;
    float dx = 1;

    laplacian(Nx,Nz,dx,dz);
    return 0;
}
